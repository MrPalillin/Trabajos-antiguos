#include "hip/hip_runtime.h"
/*
 * Simulacion simplificada de bombardeo de particulas de alta energia
 *
 * Computacion Paralela (Grado en Informatica)
 * 2017/2018
 *
 * (c) 2018 Arturo Gonzalez Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<cputils.h>

#define PI	3.14159f
#define UMBRAL	0.001f

/* Estructura para almacenar los datos de una tormenta de particulas */
typedef struct {
	int size;
	int *posval;
} Storm;

__global__ void relajacionCopia(float* layerGPU,float* layerCopyGPU,int layer_size){
	int idGlobal = threadIdx.x+(blockDim.x*threadIdx.y)+(blockDim.x*blockDim.y*blockIdx.x);
	if(idGlobal > layer_size-1) return;
	layerCopyGPU[idGlobal] = layerGPU[idGlobal];
}

__global__ void relajacionActualiza(float* layerGPU,float* layerCopyGPU,int layer_size){
	int idGlobal = threadIdx.x+(blockDim.x*threadIdx.y)+(blockDim.x*blockDim.y*blockIdx.x);
	if(idGlobal > layer_size-1) return;
	if(idGlobal != 0 && idGlobal != layer_size-1)
		layerGPU[idGlobal] = ( layerCopyGPU[idGlobal-1] + layerCopyGPU[idGlobal] + layerCopyGPU[idGlobal+1] ) / 3;
}

__global__ void copiaAtmp(float *layerGPU,float *tmp,int layer_size,int *pos,float *ini,float *fin){
	int idGlobal = threadIdx.x+(blockDim.x*threadIdx.y)+(blockDim.x*blockDim.y*blockIdx.x);
	if(idGlobal > layer_size-1) return;
	if(idGlobal != 0 && idGlobal != layer_size-1){
		tmp[idGlobal-1] = layerGPU[idGlobal];
		pos[idGlobal-1] = idGlobal;
	}
	if(idGlobal == 0){
		ini[0] = layerGPU[0];
		fin[0] = layerGPU[layer_size-1];

		if(tmp[0] <= ini[0])
			tmp[0] = 0.0f;

		if(tmp[layer_size-3] <= fin[0])
			tmp[layer_size-3] = 0.0f;
	}

	if(idGlobal > 0 && idGlobal < layer_size-3){
		if(tmp[idGlobal] == tmp[idGlobal+1]){
			tmp[idGlobal] = 0.0f;
			tmp[idGlobal+1] = 0.0f;
		}
		if(tmp[idGlobal] == tmp[idGlobal-1]){
			tmp[idGlobal] = 0.0f;
			tmp[idGlobal-1] = 0.0f;
		}
	}


}

__global__ void reduccion(float *layerGPU,float *maximosGPU,int i,int layer_size,float *tmp,int tam,int *pos){
	int idGlobal = threadIdx.x+(blockDim.x*threadIdx.y)+(blockDim.x*blockDim.y*blockIdx.x);
	if(idGlobal > (tam/2)-1) return;
		if(tmp[idGlobal] < tmp[idGlobal+(tam/2)]){
			tmp[idGlobal] = tmp[idGlobal+(tam/2)];
			pos[idGlobal] = pos[idGlobal+(tam/2)];
		}else if(tmp[idGlobal] == tmp[idGlobal+(tam/2)] && pos[idGlobal] > pos[idGlobal+(tam/2)]){
			pos[idGlobal] = pos[idGlobal+(tam/2)];
		}

	if(tam%2 != 0 && idGlobal == 0 && tmp[0] < tmp[tam-1]){
		tmp[0] = tmp[tam-1];
		pos[0] = pos[tam-1];
	}
}

__global__ void copiaAarray(float *maximosGPU,int *posicionesGPU,int i,float *tmp,int *pos,float *ini,float *fin,int layer_size,float *layerGPU){
	maximosGPU[i] = tmp[0];
	posicionesGPU[i] = pos[0];
	if(tmp[0] == 0.000000){
		posicionesGPU[i] = 0;
	}
}

__global__ void vaciarTemporales(float *tmp,int *pos,float *ini,float *fin,int layer_size){
	int idGlobal = threadIdx.x+(blockDim.x*threadIdx.y)+(blockDim.x*blockDim.y*blockIdx.x);
	if(idGlobal > layer_size-3) return;
	tmp[idGlobal] = 0;
	pos[idGlobal] = idGlobal;
	if(idGlobal == 0){
		ini[0] = 0.0f;
		fin[0] = 0.0f;
	}
}

/* ESTA FUNCION PUEDE SER MODIFICADA */
/* Funcion para actualizar una posicion de la capa */
__global__ void actualiza( float *layer, int pos, float energia,int layer_size ) {
	int idGlobal = threadIdx.x+(blockDim.x*threadIdx.y)+(blockDim.x*blockDim.y*blockIdx.x);
	if(idGlobal > layer_size-1) return;
	int distancia = pos - idGlobal;
	if ( distancia < 0 ) distancia = - distancia;

	distancia = distancia + 1;

	float atenuacion = sqrtf( (float)distancia );

	float energia_k = energia / atenuacion;

	if ( energia_k >= UMBRAL || energia_k <= -UMBRAL )
		layer[idGlobal] = layer[idGlobal] + energia_k;
}


/* FUNCIONES AUXILIARES: No se utilizan dentro de la medida de tiempo, dejar como estan */
/* Funcion de DEBUG: Imprimir el estado de la capa */
void debug_print(int layer_size, float *layer, int *posiciones, float *maximos, int num_storms ) {
	int i,k;
	if ( layer_size <= 35 ) {
		/* Recorrer capa */
		for( k=0; k<layer_size; k++ ) {
			/* Escribir valor del punto */
			printf("%10.4f |", layer[k] );

			/* Calcular el numero de caracteres normalizado con el maximo a 60 */
			int ticks = (int)( 60 * layer[k] / maximos[num_storms-1] );

			/* Escribir todos los caracteres menos el ultimo */
			for (i=0; i<ticks-1; i++ ) printf("o");

			/* Para maximos locales escribir ultimo caracter especial */
			if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
				printf("x");
			else
				printf("o");

			/* Si el punto es uno de los maximos especiales, annadir marca */
			for (i=0; i<num_storms; i++) 
				if ( posiciones[i] == k ) printf(" M%d", i );

			/* Fin de linea */
			printf("\n");
		}
	}
}

/*
 * Funcion: Lectura de fichero con datos de tormenta de particulas
 */
Storm read_storm_file( char *fname ) {
	FILE *fstorm = cp_abrir_fichero( fname );
	if ( fstorm == NULL ) {
		fprintf(stderr,"Error: Opening storm file %s\n", fname );
		exit( EXIT_FAILURE );
	}

	Storm storm;	
	int ok = fscanf(fstorm, "%d", &(storm.size) );
	if ( ok != 1 ) {
		fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
		exit( EXIT_FAILURE );
	}

	storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
	if ( storm.posval == NULL ) {
		fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
		exit( EXIT_FAILURE );
	}
	
	int elem;
	for ( elem=0; elem<storm.size; elem++ ) {
		ok = fscanf(fstorm, "%d %d\n", 
					&(storm.posval[elem*2]),
					&(storm.posval[elem*2+1]) );
		if ( ok != 2 ) {
			fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
			exit( EXIT_FAILURE );
		}
	}
	fclose( fstorm );

	return storm;
}

/*
 * PROGRAMA PRINCIPAL
 */
int main(int argc, char *argv[]) {
	int i,j,k;

	/* 1.1. Leer argumentos */
	if (argc<3) {
		fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
		exit( EXIT_FAILURE );
	}

	int layer_size = atoi( argv[1] );
	int num_storms = argc-2;
	Storm storms[ num_storms ];

	/* 1.2. Leer datos de storms */
	for( i=2; i<argc; i++ ) 
		storms[i-2] = read_storm_file( argv[i] );

	/* 1.3. Inicializar maximos a cero */
	float maximos[ num_storms ];
	int posiciones[ num_storms ];
	for (i=0; i<num_storms; i++) {
		maximos[i] = 0.0f;
		posiciones[i] = 0;
	}

	/* 2. Inicia medida de tiempo */
	hipSetDevice(0);
	hipDeviceSynchronize();
	double ttotal = cp_Wtime();

	/* COMIENZO: No optimizar/paralelizar el main por encima de este punto */

	/* 3. Reservar memoria para las capas e inicializar a cero */
	float *layer = (float *)malloc( sizeof(float) * layer_size );
	float *layer_copy = (float *)malloc( sizeof(float) * layer_size );
	if ( layer == NULL || layer_copy == NULL ) {
		fprintf(stderr,"Error: Allocating the layer memory\n");
		exit( EXIT_FAILURE );
	}
	for( k=0; k<layer_size; k++ ) layer[k] = 0.0f;
	for( k=0; k<layer_size; k++ ) layer_copy[k] = 0.0f;

	float *layerGPU;
	float *layerCopyGPU;
	float *maximosGPU;
	int *posicionesGPU;
	float* ini;
	float* fin;

	hipError_t errorLayerGPU = hipMalloc(&layerGPU,sizeof(float)*layer_size);
	hipError_t errorLayerCopyGPU = hipMalloc(&layerCopyGPU,sizeof(float)*layer_size);
	hipError_t errorMaxGPU = hipMalloc(&maximosGPU,sizeof(float)*num_storms);
	hipError_t errorPosGPU = hipMalloc(&posicionesGPU,sizeof(int)*num_storms);

	hipError_t errorIni = hipMalloc(&ini,sizeof(float));
	hipError_t errorFin = hipMalloc(&fin,sizeof(float));

	dim3 numThreads(8,32);
	int threads_per_block=256;

	int numBlocks;

	if(layer_size <= threads_per_block){
		numBlocks = 1;
	}else{
		numBlocks = layer_size/threads_per_block;
		if(layer_size%threads_per_block != 0)
			numBlocks++;
	}

	float *tmpGPU;
	int *posGPU;

	hipError_t errorTmp = hipMalloc(&tmpGPU,sizeof(float)*layer_size-2);
	hipError_t errorPos = hipMalloc(&posGPU,sizeof(int)*layer_size-2);

	float energia;
	int posicion;

	for( i=0; i<num_storms; i++) {

		//¿Como dividir los indices?
		//¿Como usar el storms en el device?

		for( j=0; j<storms[i].size; j++ ) {
			energia = (float)storms[i].posval[j*2+1] / 1000;
			posicion = storms[i].posval[j*2];
			actualiza<<<numBlocks,numThreads>>>(layerGPU,posicion,energia,layer_size);			
		}

		relajacionCopia<<<numBlocks,numThreads>>>(layerGPU,layerCopyGPU,layer_size);
		
		relajacionActualiza<<<numBlocks,numThreads>>>(layerGPU,layerCopyGPU,layer_size);

		copiaAtmp<<<numBlocks,numThreads>>>(layerGPU,tmpGPU,layer_size,posGPU,ini,fin);

		for(int tam = layer_size-2; tam > 1; tam = tam/2){
			reduccion<<<numBlocks,numThreads>>>(layerGPU,maximosGPU,i,layer_size,tmpGPU,tam,posGPU);
		}

		copiaAarray<<<1,1>>>(maximosGPU,posicionesGPU,i,tmpGPU,posGPU,ini,fin,layer_size,layerGPU);

		vaciarTemporales<<<numBlocks,numThreads>>>(tmpGPU,posGPU,ini,fin,layer_size);

	}

	hipError_t errorHostCpyLayer = hipMemcpy(layer,layerGPU,sizeof(float)*layer_size,hipMemcpyDeviceToHost);
	hipError_t errorHostCpyMax = hipMemcpy(maximos,maximosGPU,sizeof(float)*num_storms,hipMemcpyDeviceToHost);
	hipError_t errorHostCpyPos = hipMemcpy(posiciones,posicionesGPU,sizeof(int)*num_storms,hipMemcpyDeviceToHost);

	hipError_t freeLayer = hipFree(layerGPU);
	hipError_t freeLayerCopy = hipFree(layerCopyGPU);
	hipError_t freeMax = hipFree(maximosGPU);
	hipError_t freePos = hipFree(posicionesGPU);

	hipError_t freeTmp = hipFree(tmpGPU);
	hipError_t freePosTmp = hipFree(posGPU);
	hipError_t freeIni = hipFree(ini);
	hipError_t freeFin = hipFree(fin);

	/* FINAL: No optimizar/paralelizar por debajo de este punto */

	/* 6. Final de medida de tiempo */
	hipDeviceSynchronize();
	ttotal = cp_Wtime() - ttotal;

	/* 7. DEBUG: Dibujar resultado (Solo para capas con hasta 35 puntos) */
	#ifdef DEBUG
	debug_print( layer_size, layer, posiciones, maximos, num_storms );
	#endif

	/* 8. Salida de resultados para tablon */
	printf("\n");
	/* 8.1. Tiempo total de la computacion */
	printf("Time: %lf\n", ttotal );
	/* 8.2. Escribir los maximos */
	printf("Result:");
	for (i=0; i<num_storms; i++)
		printf(" %d %f", posiciones[i], maximos[i] );
	printf("\n");

	/* 9. Liberar recursos */	
	for( i=0; i<argc-2; i++ )
		free( storms[i].posval );

	/* 10. Final correcto */
	return 0;
}
